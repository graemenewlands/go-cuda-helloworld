
#include <hip/hip_runtime.h>
// #include <stdio.h>

__global__ void sumReduction(int *input, int *output, int n) {
    int tid = threadIdx.x;
    extern __shared__ int sdata[];

    // Load shared mem from global mem
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = (i < n) ? input[i] : 0;
    __syncthreads();

    // Do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global mem
    if (tid == 0) output[blockIdx.x] = sdata[0];
}

extern "C" {
    int test_redux(int size, int blockSize) {
        int *input, *d_input, *d_partial_sums, *partial_sums;
        int numBlocks = (size + blockSize - 1) / blockSize;
        int smemSize = blockSize * sizeof(int); // Size of shared memory per block

        // Allocate host memory
        input = (int*)malloc(size * sizeof(int));
        partial_sums = (int*)malloc(numBlocks * sizeof(int));

        // Initialize input data
        for (int i = 0; i < size; i++) {
            input[i] = 1; // Example data
        }

        // Allocate device memory
        hipMalloc(&d_input, size * sizeof(int));
        hipMalloc(&d_partial_sums, numBlocks * sizeof(int));

        // Copy data from host to device
        hipMemcpy(d_input, input, size * sizeof(int), hipMemcpyHostToDevice);

        // Launch kernel
        sumReduction<<<numBlocks, blockSize, smemSize>>>(d_input, d_partial_sums, size);

        // Copy partial sums from device to host
        hipMemcpy(partial_sums, d_partial_sums, numBlocks * sizeof(int), hipMemcpyDeviceToHost);

        // Finish reduction on the host
        int totalSum = 0;
        for (int i = 0; i < numBlocks; i++) {
            totalSum += partial_sums[i];
        }

        // printf("Total sum is %d\n", totalSum);

        // Cleanup
        hipFree(d_input);
        hipFree(d_partial_sums);
        free(input);
        free(partial_sums);

        return totalSum;
    }

}